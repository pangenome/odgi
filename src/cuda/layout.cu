#include "hip/hip_runtime.h"
#include "layout.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include "hip/hip_runtime_api.h"
#include <openrand/philox.h>


#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

namespace cuda {


__device__ double compute_zeta(uint32_t n, double theta) {
    double ans = 0.0;
    for (uint32_t i = 1; i <= n; i++) {
        ans += pow(1.0 / double(i), theta);
    }
    return ans;
}

// this function uses the cuda operation __powf, which is a faster but less precise alternative to the pow operation
__device__ uint32_t cuda_rnd_zipf(openrand::Philox &rnd_state, uint32_t n, double theta, double zeta2, double zetan) {
    double alpha = 1.0 / (1.0 - theta);
    double denominator = 1.0 - zeta2 / zetan;
    if (denominator == 0.0) {
        denominator = 1e-9;
    }
    double eta = (1.0 - __powf(2.0 / double(n), 1.0 - theta)) / (denominator);

    // INFO: hiprand_uniform generates random values between 0.0 (excluded) and 1.0 (included)
    double u = 1.0 - rnd_state.rand<float>();;
    double uz = u * zetan;

    int64_t val = 0;
    if (uz < 1.0) val = 1;
    else if (uz < 1.0 + __powf(0.5, theta)) val = 2;
    else val = 1 + int64_t(double(n) * __powf(eta * u - eta + 1.0, alpha));

    if (val > n) {
        //printf("WARNING: val: %ld, n: %u\n", val, uint32_t(n));
        val--;
    }
    assert(val >= 0);
    assert(val <= n);
    return uint32_t(val);
}


/**
* @brief: update the coordinates of two visualization nodes in the 2D layout space
* This function is called multiple times in one `gpu_layout_kernel` in order to increase the data reuse. 
* Each time, the warp shuffle intrinsics are used to change the selection of node 2 among the 32 threads in the warp. 
* E.g. Iter : Step Pairs Selected would be: 
*     1: (a0, b0), (a1, b1), (a2, b2), ..., (a31, b31)
*     2: (a0, b9), (a1, b0), (a2, b3), ..., (a31, b4)
*     3: (a0, b1), (a1, b4), (a2, b1), ..., (a31, b10)
*     ...
* `b` is randomly chosen from the 32 threads in the warp. 
* @param n1_pos_in_path: position of node 1 in the current selected path
* @param n1_id: id of node 1
* @param n1_offset: offset of node 1
* @param n2_pos_in_path: position of node 2 in the current selected path
* @param n2_id: id of node 2
* @param n2_offset: offset of node 2
* @param eta: an coefficient used in the update formula
* @param node_data: the data structure that stores the coordinates of all nodes
*/
__device__
void update_pos_gpu(int64_t &n1_pos_in_path, uint32_t &n1_id, int &n1_offset,
                    int64_t &n2_pos_in_path, uint32_t &n2_id, int &n2_offset,
                    double eta, 
                    cuda::node_data_t &node_data) {
    double term_dist = std::abs(static_cast<double>(n1_pos_in_path) - static_cast<double>(n2_pos_in_path));

    if (term_dist < 1e-9) {
        term_dist = 1e-9;
    }

    double w_ij = 1.0 / term_dist;

    double mu = eta * w_ij;
    if (mu > 1.0) {
        mu = 1.0;
    }

    float *x1 = &node_data.nodes[n1_id].coords[n1_offset];
    float *x2 = &node_data.nodes[n2_id].coords[n2_offset];
    float *y1 = &node_data.nodes[n1_id].coords[n1_offset + 1];
    float *y2 = &node_data.nodes[n2_id].coords[n2_offset + 1];
    double x1_val = double(*x1);
    double x2_val = double(*x2);
    double y1_val = double(*y1);
    double y2_val = double(*y2);

    double dx = x1_val - x2_val;
    double dy = y1_val - y2_val;

    if (dx == 0.0) {
        dx = 1e-9;
    }

    double mag = sqrt(dx * dx + dy * dy);
    double delta = mu * (mag - term_dist) / 2.0;
    //double delta_abs = std::abs(delta);

    // TODO implement delta max stop functionality
    double r = delta / mag;
    double r_x = r * dx;
    double r_y = r * dy;
    // TODO check current value before updating
    atomicExch(x1, float(x1_val - r_x));
    atomicExch(x2, float(x2_val + r_x));
    atomicExch(y1, float(y1_val - r_y));
    atomicExch(y2, float(y2_val + r_y)); 
}

__global__ 
void gpu_layout_kernel(int iter, cuda::layout_config_t config, double eta, double *zetas, 
                                   cuda::node_data_t node_data, cuda::path_data_t path_data, int sm_count) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    //curandStateCoalesced_t *thread_rnd_state = &rnd_state[smid];
    openrand::Philox rng(tid, iter); // unique stream for each thread at each iteration
    
    // need upto 4 coin flips per thread, this has 32 bits, 32 flips
    const int coin_flips = rng.rand<int>();

    const bool flip1 = coin_flips & 1;
    const bool flip2 = coin_flips & 2;
    const bool flip3 = coin_flips & 4;
    const bool flip4 = coin_flips & 8;

    __shared__ bool cooling[BLOCK_SIZE / WARP_SIZE]; 
    if (threadIdx.x % WARP_SIZE == 1) {
        cooling[threadIdx.x / WARP_SIZE] = (iter >= config.first_cooling_iteration) || flip1;
    }

    // select path
    __shared__ uint32_t first_step_idx[BLOCK_SIZE / WARP_SIZE]; // BLOCK_SIZE/WARP_SIZE = 1024/32 = 32
    // each thread picks its own path
    uint32_t step_idx = rng.uniform<int>(0, path_data.total_path_steps);

    uint32_t path_idx = path_data.element_array[step_idx].pidx;
    path_t p = path_data.paths[path_idx];

    if (p.step_count < 2) {
        return;
    }
    assert(p.step_count > 1);

    // INFO: hiprand_uniform generates random values between 0.0 (excluded) and 1.0 (included)
    uint32_t s1_idx = rng.uniform<int>(0, p.step_count);
    assert(s1_idx < p.step_count);
    uint32_t s2_idx;

    if (cooling[threadIdx.x / WARP_SIZE]) {
        bool backward;
        uint32_t jump_space;
        if (s1_idx > 0 && flip2 || s1_idx == p.step_count-1) {
            // go backward
            backward = true;
            jump_space = min(config.space, s1_idx);
        } else {
            // go forward
            backward = false;
            jump_space = min(config.space, p.step_count - s1_idx - 1);
        }
        uint32_t space = jump_space;
        if (jump_space > config.space_max) {
            space = config.space_max + (jump_space - config.space_max) / config.space_quantization_step + 1;
        }

        uint32_t z_i = cuda_rnd_zipf(rng, jump_space, config.theta, zetas[2], zetas[space]);

        s2_idx = backward ? s1_idx - z_i : s1_idx + z_i;
    } else {
        do {
            s2_idx = rng.uniform<int>(0, p.step_count);
        } while (s1_idx == s2_idx);
    }
    assert(s1_idx < p.step_count);
    assert(s2_idx < p.step_count);
    assert(s1_idx != s2_idx);


    uint32_t n1_id = p.elements[s1_idx].node_id;
    int64_t n1_pos_in_path = p.elements[s1_idx].pos;
    bool n1_is_rev = (n1_pos_in_path < 0)? true: false;
    n1_pos_in_path = std::abs(n1_pos_in_path);

    uint32_t n2_id = p.elements[s2_idx].node_id;
    int64_t n2_pos_in_path = p.elements[s2_idx].pos;
    bool n2_is_rev = (n2_pos_in_path < 0)? true: false;
    n2_pos_in_path = std::abs(n2_pos_in_path);

    uint32_t n1_seq_length = node_data.nodes[n1_id].seq_length;
    bool n1_use_other_end = flip3 ? true : false;
    if (n1_use_other_end) {
        n1_pos_in_path += uint64_t{n1_seq_length};
        n1_use_other_end = !n1_is_rev;
    } else {
        n1_use_other_end = n1_is_rev;
    }

    uint32_t n2_seq_length = node_data.nodes[n2_id].seq_length;
    bool n2_use_other_end = flip4 ? true : false;
    if (n2_use_other_end) {
        n2_pos_in_path += uint64_t{n2_seq_length};
        n2_use_other_end = !n2_is_rev;
    } else {
        n2_use_other_end = n2_is_rev;
    }

    int n1_offset = n1_use_other_end? 2: 0;
    int n2_offset = n2_use_other_end? 2: 0;

    // Update Coordinates based on the data of selected nodes: n_pos_in_path, n_id, n_offset
    update_pos_gpu(n1_pos_in_path, n1_id, n1_offset, 
                   n2_pos_in_path, n2_id, n2_offset, 
                   eta, node_data);
}


void gpu_layout(layout_config_t config, const odgi::graph_t &graph, std::vector<std::atomic<double>> &X, std::vector<std::atomic<double>> &Y) {


    std::cout << "===== Use GPU to compute odgi-layout =====" << std::endl;
    // get cuda device property, and get the SM count
    hipDeviceProp_t prop;
    CUDACHECK(hipGetDeviceProperties(&prop, 0));
    int sm_count = prop.multiProcessorCount;

    // create eta array
    double *etas;
    hipMallocManaged(&etas, config.iter_max * sizeof(double));

    const int32_t iter_max = config.iter_max;
    const int32_t iter_with_max_learning_rate = config.iter_with_max_learning_rate;
    const double w_max = 1.0;
    const double eps = config.eps;
    const double eta_max = config.eta_max;
    const double eta_min = eps / w_max;
    const double lambda = log(eta_max / eta_min) / ((double) iter_max - 1);
    for (int32_t i = 0; i < config.iter_max; i++) {
        double eta = eta_max * exp(-lambda * (std::abs(i - iter_with_max_learning_rate)));
        etas[i] = isnan(eta)? eta_min : eta;
    }

    // create node data structure
    // consisting of sequence length and coords
    uint32_t node_count = graph.get_node_count();
    assert(graph.min_node_id() == 1);
    assert(graph.max_node_id() == node_count);
    assert(graph.max_node_id() - graph.min_node_id() + 1 == node_count);

    cuda::node_data_t node_data;
    node_data.node_count = node_count;
    hipMallocManaged(&node_data.nodes, node_count * sizeof(cuda::node_t));
    for (int node_idx = 0; node_idx < node_count; node_idx++) {
        //assert(graph.has_node(node_idx));
        cuda::node_t *n_tmp = &node_data.nodes[node_idx];

        // sequence length
        const handlegraph::handle_t h = graph.get_handle(node_idx + 1, false);
        // NOTE: unable store orientation (reverse), since this information is path dependent
        n_tmp->seq_length = graph.get_length(h);

        // copy random coordinates
        n_tmp->coords[0] = float(X[node_idx * 2].load());
        n_tmp->coords[1] = float(Y[node_idx * 2].load());
        n_tmp->coords[2] = float(X[node_idx * 2 + 1].load());
        n_tmp->coords[3] = float(Y[node_idx * 2 + 1].load());
    }


    // create path data structure
    uint32_t path_count = graph.get_path_count();
    cuda::path_data_t path_data;
    path_data.path_count = path_count;
    path_data.total_path_steps = 0;
    hipMallocManaged(&path_data.paths, path_count * sizeof(cuda::path_t));

    vector<odgi::path_handle_t> path_handles{};
    path_handles.reserve(path_count);
    graph.for_each_path_handle(
        [&] (const odgi::path_handle_t& p) {
            path_handles.push_back(p);
            path_data.total_path_steps += graph.get_step_count(p);
        });
    hipMallocManaged(&path_data.element_array, path_data.total_path_steps * sizeof(path_element_t));

    // get length and starting position of all paths
    uint64_t first_step_counter = 0;
    for (int path_idx = 0; path_idx < path_count; path_idx++) {
        odgi::path_handle_t p = path_handles[path_idx];
        int step_count = graph.get_step_count(p);
        path_data.paths[path_idx].step_count = step_count;
        path_data.paths[path_idx].first_step_in_path = first_step_counter;
        first_step_counter += step_count;
    }

#pragma omp parallel for num_threads(config.nthreads)
    for (int path_idx = 0; path_idx < path_count; path_idx++) {
        odgi::path_handle_t p = path_handles[path_idx];
        //std::cout << graph.get_path_name(p) << ": " << graph.get_step_count(p) << std::endl;

        uint32_t step_count = path_data.paths[path_idx].step_count;
        uint64_t first_step_in_path = path_data.paths[path_idx].first_step_in_path;
        if (step_count == 0) {
            path_data.paths[path_idx].elements = NULL;
        } else {
            path_element_t *cur_path = &path_data.element_array[first_step_in_path];
            path_data.paths[path_idx].elements = cur_path;

            odgi::step_handle_t s = graph.path_begin(p);
            int64_t pos = 1;
            // Iterate through path
            for (int step_idx = 0; step_idx < step_count; step_idx++) {
                odgi::handle_t h = graph.get_handle_of_step(s);
                //std::cout << graph.get_id(h) << std::endl;

                cur_path[step_idx].node_id = graph.get_id(h) - 1;
                cur_path[step_idx].pidx = uint32_t(path_idx);
                // store position negative when handle reverse
                if (graph.get_is_reverse(h)) {
                    cur_path[step_idx].pos = -pos;
                } else {
                    cur_path[step_idx].pos = pos;
                }
                pos += graph.get_length(h);

                // get next step
                if (graph.has_next_step(s)) {
                    s = graph.get_next_step(s);
                } else if (!(step_idx == step_count-1)) {
                    // should never be reached
                    std::cout << "Error: Here should be another step" << std::endl;
                }
            }
        }
    }

    // cache zipf zetas
    auto start_zeta = std::chrono::high_resolution_clock::now();
    double *zetas;
    uint64_t zetas_cnt = ((config.space <= config.space_max)? config.space : (config.space_max + (config.space - config.space_max) / config.space_quantization_step + 1)) + 1;
    hipMallocManaged(&zetas, zetas_cnt * sizeof(double));
    double zeta_tmp = 0.0;
    for (uint64_t i = 1; i < config.space + 1; i++) {
        zeta_tmp += dirtyzipf::fast_precise_pow(1.0 / i, config.theta);
        if (i <= config.space_max) {
            zetas[i] = zeta_tmp;
        }
        if (i >= config.space_max && (i - config.space_max) % config.space_quantization_step == 0) {
            zetas[config.space_max + 1 + (i - config.space_max) / config.space_quantization_step] = zeta_tmp;
        }
    }
    auto end_zeta = std::chrono::high_resolution_clock::now();
    uint32_t duration_zeta_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_zeta - start_zeta).count();

    const uint64_t block_size = BLOCK_SIZE;
    uint64_t block_nbr = (config.min_term_updates + block_size - 1) / block_size; 

    // hiprandState_t *rnd_state_tmp;
    // curandStateCoalesced_t *rnd_state;
    // CUDACHECK(hipMallocManaged(&rnd_state_tmp, sm_count * block_size * sizeof(hiprandState_t)));
    // CUDACHECK(hipMallocManaged(&rnd_state, sm_count * sizeof(curandStateCoalesced_t)));
    // cuda_device_init<<<sm_count, block_size>>>(rnd_state_tmp, rnd_state);
    // CUDACHECK(hipGetLastError());
    // CUDACHECK(hipDeviceSynchronize());
    // hipFree(rnd_state_tmp);

    // one curandStateCoalesced_t for each sm, not each block. So several blocks 
    // share the same curandStateCoalesced_t.

    for (int iter = 0; iter < config.iter_max; iter++) {
        gpu_layout_kernel<<<block_nbr, block_size>>>(iter, config, etas[iter], zetas, node_data, path_data, sm_count);
        // check error
        CUDACHECK(hipGetLastError());
        CUDACHECK(hipDeviceSynchronize());
    }

    // copy coords back to X, Y vectors
    for (int node_idx = 0; node_idx < node_count; node_idx++) {
        cuda::node_t *n = &(node_data.nodes[node_idx]);
        // coords[0], coords[1], coords[2], coords[3] are stored consecutively. 
        float *coords = n->coords;
        // check if coordinates valid (not NaN or infinite)
        for (int i = 0; i < 4; i++) {
            if (!isfinite(coords[i])) {
                std::cout << "WARNING: invalid coordiate" << std::endl;
            }
        }
        X[node_idx * 2].store(double(coords[0]));
        Y[node_idx * 2].store(double(coords[1]));
        X[node_idx * 2 + 1].store(double(coords[2]));
        Y[node_idx * 2 + 1].store(double(coords[3]));
        //std::cout << "coords of " << node_idx << ": [" << X[node_idx*2] << "; " << Y[node_idx*2] << "] ; [" << X[node_idx*2+1] << "; " << Y[node_idx*2+1] <<"]\n";
    }

    // free memory
    hipFree(etas);
    hipFree(node_data.nodes);
    hipFree(path_data.paths);
    hipFree(path_data.element_array);
    hipFree(zetas);
    // hipFree(rnd_state);

    return;
}

}